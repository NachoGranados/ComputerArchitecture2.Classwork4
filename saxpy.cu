#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

//CPU function
void saxpy_h(int n, float a, float *x, float *y){
    for(int i = 0; i < n; i++)
        y[i] = a*x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements 
  clock_t start_d=clock();
  printf("Doing GPU Vector SAXPY\n");
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  clock_t end_d = clock();

    printf("Doing CPU Vector add\n");
    clock_t start_h = clock();
    saxpy_h(N, 2.0f, x, y);
    clock_t end_h = clock();

    //Time computing
    double time_d = (double)(end_d-start_d)/CLOCKS_PER_SEC;
    double time_h = (double)(end_h-start_h)/CLOCKS_PER_SEC;

    //Copying data back to host, this is a blocking call and will not start until all kernels are finished
    printf("\t GPU time = %f \t CPU time = %f\n", time_d, time_h);



  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}
